#include "hip/hip_runtime.h"
/*
 * SPDX-FileCopyrightText: Copyright (c) 2025 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: Apache-2.0
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <doca_gpunetio_dev_buf.cuh>
#include <doca_gpunetio_dev_rdma.cuh>

#include "doca_backend.h"

// __device__ void prepXferDoca(void *backendHandler) {
//     struct docaXferReqGpu *treq = (struct docaXferReqGpu *) backendHandler;
//     doca_error_t result;
// 	struct doca_gpu_buf *lbuf;
// 	struct doca_gpu_buf *rbuf;
//     const int connection_index = 0;
//     uint32_t num_ops=0;

//     if (threadIdx.x >= treq->num)
//         return;

//     doca_gpu_dev_buf_get_buf((struct doca_gpu_buf_arr *)treq->larr[threadIdx.x], 0, &lbuf);
//     doca_gpu_dev_buf_get_buf((struct doca_gpu_buf_arr *)treq->rarr[threadIdx.x], 0, &rbuf);

//     // printf(">>>>>>> CUDA rdma write kernel thread %d pos %d size %d\n", threadIdx.x, pos, (int)treq->size[threadIdx.x]);

//     //Each thread should send a different buffer
//     result = doca_gpu_dev_rdma_write_strong(treq->rdma_gpu, connection_index, rbuf, 0, lbuf, 0, treq->size[threadIdx.x], 0, DOCA_GPU_RDMA_WRITE_FLAG_NONE);
//     if (result != DOCA_SUCCESS)
//         printf("Error %d doca_gpu_dev_rdma_write_strong\n", result);

//     __syncthreads();

//     if (threadIdx.x == 0) {
//         result = doca_gpu_dev_rdma_commit_strong(treq->rdma_gpu, connection_index);
//         if (result != DOCA_SUCCESS)
//             printf("Error %d doca_gpu_dev_rdma_push\n", result);

//         result = doca_gpu_dev_rdma_wait_all(treq->rdma_gpu, &num_ops);
//         if (result != DOCA_SUCCESS)
//             printf("Error %d doca_gpu_dev_rdma_wait_all\n", result);

//         // printf(">>>>>>> CUDA rdma write kernel pos %d num %d completed %d ops\n", pos, treq->num, num_ops);

//         treq->num = 0;
//     }
// }

// __device__ prepXferGpu f_ptr = prepXferDoca;

// prepXferGpu postXferGpuGet()
// {
//     prepXferGpu h_prepXferGpu;

//     if (hipSuccess != hipMemcpyFromSymbol(&h_prepXferGpu, HIP_SYMBOL(f_ptr), sizeof (prepXferGpu)))
//         printf ("FAILED to get SYMBOL\n");

//     return h_prepXferGpu;
// }


__global__ void kernel_write(struct doca_gpu_dev_rdma *rdma_gpu, struct docaXferReqGpu *xferReqRing, uint32_t pos)
{
    doca_error_t result;
	struct doca_gpu_buf *lbuf;
	struct doca_gpu_buf *rbuf;
    const int connection_index = 0;
    uint32_t num_ops=0;

    //Warmup
    if (xferReqRing == nullptr)
        return;

    if (threadIdx.x >= xferReqRing[pos].num)
        return;

    doca_gpu_dev_buf_get_buf((struct doca_gpu_buf_arr *)xferReqRing[pos].larr[threadIdx.x], 0, &lbuf);
	doca_gpu_dev_buf_get_buf((struct doca_gpu_buf_arr *)xferReqRing[pos].rarr[threadIdx.x], 0, &rbuf);

    // printf(">>>>>>> CUDA rdma write kernel thread %d pos %d size %d\n", threadIdx.x, pos, (int)xferReqRing[pos].size[threadIdx.x]);

    //Each thread should send a different buffer
    result = doca_gpu_dev_rdma_write_strong(rdma_gpu, connection_index, rbuf, 0, lbuf, 0, xferReqRing[pos].size[threadIdx.x], 0, DOCA_GPU_RDMA_WRITE_FLAG_NONE);
    if (result != DOCA_SUCCESS)
        printf("Error %d doca_gpu_dev_rdma_write_strong\n", result);

    __syncthreads();

    if (threadIdx.x == 0) {
        result = doca_gpu_dev_rdma_commit_strong(rdma_gpu, connection_index);
        if (result != DOCA_SUCCESS)
            printf("Error %d doca_gpu_dev_rdma_push\n", result);

        result = doca_gpu_dev_rdma_wait_all(rdma_gpu, &num_ops);
        if (result != DOCA_SUCCESS)
            printf("Error %d doca_gpu_dev_rdma_wait_all\n", result);

        // printf(">>>>>>> CUDA rdma write kernel pos %d num %d completed %d ops\n", pos, xferReqRing[pos].num, num_ops);

        xferReqRing[pos].num = 0;
    }

}

extern "C" {

doca_error_t doca_kernel_write(hipStream_t stream, struct doca_gpu_dev_rdma *rdma_gpu, struct docaXferReqGpu *xferReqRing, uint32_t pos)
{
    hipError_t result = hipSuccess;

    if (rdma_gpu == NULL) {
        fprintf(stderr, "kernel_write_server invalid input values");
        return DOCA_ERROR_INVALID_VALUE;
    }

    /* Check no previous CUDA errors */
    result = hipGetLastError();
    if (result != hipSuccess) {
        fprintf(stderr, "[%s:%d] cuda failed with %s", __FILE__, __LINE__, hipGetErrorString(result));
        return DOCA_ERROR_BAD_STATE;
    }

    kernel_write<<<1, DOCA_XFER_REQ_SIZE, 0, stream>>>(rdma_gpu, xferReqRing, pos);
    result = hipGetLastError();
    if (result != hipSuccess) {
        fprintf(stderr, "[%s:%d] cuda failed with %s", __FILE__, __LINE__, hipGetErrorString(result));
        return DOCA_ERROR_BAD_STATE;
    }

    return DOCA_SUCCESS;
}

} /* extern C */
